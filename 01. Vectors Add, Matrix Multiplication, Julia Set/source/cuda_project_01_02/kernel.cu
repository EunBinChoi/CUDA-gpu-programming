#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "opencv\cv.h"
#include "opencv\highgui.h"

//#define NUM_DATA 123456
using namespace cv;

__global__ void vec_add(float *a, float *b, float *c, int n){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid >= n) return;
	c[tid] = a[tid] + b[tid];
}

// �ǽ� 1-2
int main(){
	int data = 123456, thread; // data : �������� ����, thread : �������� ����
	hipEvent_t start,end;
	float gpu_time;

	hipEventCreate(&start);
	hipEventCreate(&end);

	float *a, *b, *c;
	a = (float *)malloc(sizeof(float) * data);
	b = (float *)malloc(sizeof(float) * data);
	c = (float *)malloc(sizeof(float) * data);

	for(int i = 0 ; i < data ; i++){
		a[i] = i*2;
		b[i] = i+3;
	}

	float *da, *db, *dc;
	hipMalloc((void **)&da, sizeof(float)*data);
	hipMalloc((void **)&db, sizeof(float)*data);
	hipMalloc((void **)&dc, sizeof(float)*data);

	hipMemcpy(da, a, sizeof(float)*data,hipMemcpyHostToDevice);
	hipMemcpy(da, b, sizeof(float)*data,hipMemcpyHostToDevice);

	int blocknum;

	for(thread = 2 ; thread <= 1024 ; thread=thread*2){
		gpu_time = 0;
		blocknum = data/thread;

		hipEventRecord(start, 0);
		vec_add<<<blocknum, thread>>>(da,db,dc,data);
		hipEventRecord(end, 0);

		hipEventSynchronize(end);
		hipEventElapsedTime(&gpu_time, start, end);
		hipMemcpy(da, c, sizeof(float)*data,hipMemcpyDeviceToHost);

		printf("���� thread ���� = %d, gpu time = %f\n", thread, gpu_time);
	}

	free(a);free(b);free(c);
	hipFree(da);hipFree(db);hipFree(dc);
	return 0;
}