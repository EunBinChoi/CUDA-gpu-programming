#include "hip/hip_runtime.h"
// 3�� ����-GPU�� CPU ��
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "opencv\cv.h"
#include "opencv\highgui.h"

using namespace cv;

typedef struct CudaComplex{ // ���Ҽ� ���� // ���� 3��
	float r;
	float i;

}Cuda_C;

__device__ __host__ void complex_add(float a, float b, float c, float d, float *realOut, float *imgOut){
	*realOut = a + c;
	*imgOut = b + d;

}

__device__ __host__ void complex_sub(float a, float b, float c, float d, float *realOut, float *imgOut){
	*realOut = a - c;
	*imgOut = b - d;

}

__device__ __host__ void complex_mul(float a, float b, float c, float d, float *realOut, float *imgOut){
	*realOut = (a*c) - (b*d);
	*imgOut = (b*c) + (a*d);

}

__device__ __host__ void complex_div(float a, float b, float c, float d, float *realOut, float *imgOut){
	*realOut = ((a*c) + (b*d)) / (pow(c,2)+pow(d,2));
	*imgOut = ((b*c) - (a*d)) / (pow(c,2)+pow(d,2));

}
__device__ __host__ Cuda_C complex_add(Cuda_C a, Cuda_C b){

	Cuda_C result;
	result.r = a.r + b.r;
	result.i = a.i + b.i;

	return result;

}
__device__ __host__ Cuda_C complex_mul(Cuda_C a, Cuda_C b){

	Cuda_C result;
	result.r = (a.r*b.r) - (a.i*b.i);
	result.i = (a.r*b.i) + (a.i*b.r);

	return result;
}
__device__ __host__ Cuda_C complex_power(Cuda_C a){

	Cuda_C result;
	result = complex_mul(a,a);
	return result;
}


__global__ void vec_add(float *a, float *b, float *c, int n){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid >= n) return;
	c[tid] = a[tid] + b[tid];

}

__global__ void mat_mul(float *a, float *b, float *c, int M, int N, int K){
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	float sum = 0;

	for(int k = 0; k < K; k++){
		//sum = 0;
		sum += a[row*K+k] * b[k*N+col];
	}

	c[row*N + col] = sum;
}

void nmat_mul(float *a, float *b, float *c, int M, int N, int K){

	float sum = 0;

	for(int i = 0 ; i < M ; i++) {
		for(int j = 0 ; j < N ; j++) {
			for(int k = 0; k < K; k++) {
				sum += a[i*K+k] * b[k*N+j];
			}
			c[i*N+j] = sum;
			sum = 0;
		}
	}
}

__global__ void gpu_julia(unsigned char* matrix, int width, int height, Cuda_C c){

	const int Mid_x = width / 2;
	const int Mid_y = height / 2;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	Cuda_C Complex_Result;
	Cuda_C Complex;

	Complex.r = (col - Mid_x) / 1024.0;
	Complex.i = (Mid_y - row) / 1024.0; 

	matrix[row*width + col] = 255;

	for(int k = 0 ; k <= 200 ; k++){
		Complex_Result = Complex;

		if((Complex_Result.r*Complex_Result.r + Complex_Result.i * Complex_Result.i) > 1000000)
		{
			matrix[row*width + col] = 0; break;
		}
		Complex_Result = complex_power(Complex);
		Complex_Result = complex_add((Complex_Result),c); 
		Complex = Complex_Result;
	}
}

int main(){
	//�ǽ� �߰� (CPU GPU ���Ѱ�)

	const int M = 2048; // x�� ����(width)
	const int N = 2048; // y�� ����(height)

	const int MidSpot_X = N/2; // x��ǥ ����
	const int MidSpot_Y = M/2; // y��ǥ ����
	unsigned char *temp;
	temp = (unsigned char *)malloc(sizeof(unsigned char) * M*N);
	
	for(int i = 0; i < M*N ; i++)
		temp[i] = 255;


	unsigned char *dMat;
	hipMalloc((void**)&dMat, sizeof(unsigned char) * M*N);

	hipMemcpy(dMat, temp, sizeof(unsigned char)*M*N,hipMemcpyHostToDevice);

	int thread;
	
	Cuda_C Complex_Result;
	Cuda_C C;

	printf("���Ҽ� C�� �Ǽ���, ����θ� �����Ͻÿ�(r,i) : ");
	scanf("%f,%f", &C.r, &C.i);
	printf("thread�� ������ �Է��ϼ��� : ");
	scanf("%d", &thread);
	
	Cuda_C Complex;

	Mat mtGray = Mat( M, N, CV_8UC1, Scalar(0));//CPU �׷��� �̹���
	Mat mtGray_cuda = Mat( M, N, CV_8UC1, Scalar(0));//GPU �׷��� �̹���

	//gpu_time = 0;
	//hipEventRecord(start, 0);

	dim3 dim_blocks(thread, thread, 1);
	dim3 dim_grid(N/thread, M/thread, 1);

	gpu_julia <<< dim_grid, dim_blocks>>>(dMat,M,N,C);
	//hipEventRecord(end, 0);
	//hipEventSynchronize(end);
	//hipEventElapsedTime(&gpu_time, start, end);
	// printf("%f\n", gpu_time);

	hipMemcpy(temp, dMat, sizeof(unsigned char)*M*N,hipMemcpyDeviceToHost);
	//temp�� �����ϴ� ������ gpu ���α׷����� ���� dMat�� �ޱ� ����

	for(int i = 0 ; i < M; i ++){
		for(int j = 0 ; j < N ; j++){
			//printf("%d",temp[i*N + j]);
			mtGray_cuda.at<unsigned char>(i,j) = temp[i*N + j];
		}   
	}
	//printf("thread = %d, gpu time = %.10f\n",thread, gpu_time);

	for(int i = 0 ; i < M; i ++){
		for(int j = 0 ; j < N ; j++){
			Complex.r = (j - MidSpot_X) / 1024.0;   //x��
			Complex.i = (MidSpot_Y - i) / 1024.0;   //y��

			mtGray.at<unsigned char>(i,j) = 255;

			for(int k = 0 ; k <= 200 ; k++){
				Complex_Result = Complex;

				if((Complex_Result.r*Complex_Result.r + Complex_Result.i * Complex_Result.i) > 1000000)
				{

					mtGray.at<unsigned char>(i,j) = 0;
					break;
				}
				else{
					Complex_Result = complex_power(Complex);
					Complex_Result = complex_add((Complex_Result),C); 
					Complex = Complex_Result;
				}
			}
		}
	}

	imshow("Window_CPU", mtGray);
	imshow("Window_CUDA", mtGray_cuda);
	waitKey(0); // â ������ ����!

	imwrite("copy_CPU.jpg", mtGray);
	imwrite("copy_CUDA.jpg", mtGray_cuda);


	free(temp);
	hipFree(dMat);

	return 0;
}



