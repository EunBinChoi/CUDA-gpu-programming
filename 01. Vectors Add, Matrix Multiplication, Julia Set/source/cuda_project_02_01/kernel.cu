#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "opencv\cv.h"
#include "opencv\highgui.h"

//#define NUM_DATA 123456
using namespace cv;


__global__ void mat_mul(float *a, float *b, float *c, int M, int N, int K){
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	float sum = 0;

	for(int k = 0; k < K; k++){
		sum += a[row*K+k] * b[k*N+col];
	}
	c[row*N + col] = sum;
}

int main(){
	// �ǽ� 2-1
	hipEvent_t start, end;
	float gpu_time; 
	
	hipEventCreate(&start);
	hipEventCreate(&end);

	const int M = 2048;
	const int N = 1536;
	const int K = 1024;

	float *a, *b, *c;
	a = (float *)malloc(sizeof(float) * M*K);
	b = (float *)malloc(sizeof(float) * K*N);
	c = (float *)malloc(sizeof(float) * M*N);

	for(int i = 0; i < M*K ; i++)
		a[i] = (float)(rand() - RAND_MAX/2)/(RAND_MAX/2); 
	for(int i = 0; i < K*N ; i++)
		b[i] = (float)(rand() - RAND_MAX/2)/(RAND_MAX/2);
	  
	float *da, *db, *dc;
	hipMalloc((void**)&da, sizeof(float) * M*K);
	hipMalloc((void**)&db, sizeof(float) * K*N);
	hipMalloc((void**)&dc, sizeof(float) * M*N);
	
	hipMemcpy(da, a, sizeof(float)*M*K,hipMemcpyHostToDevice);
	hipMemcpy(da, b, sizeof(float)*K*N,hipMemcpyHostToDevice);
	
	int thread = 0;
	printf("block �� thread ���� �Է� : ");
	scanf("%d",&thread);
	
	hipEventRecord(start, 0);
	dim3 dim_block(thread, thread, 1);
	dim3 dim_grid(N/thread, M/thread, 1);
	mat_mul<<<dim_grid, dim_block>>>(da,db,dc,M,N,K);

	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&gpu_time, start, end); // �ð� ������ ��

	hipMemcpy(c, dc, sizeof(float)*M*N,hipMemcpyDeviceToHost);
	printf("gpu time = %f\n", gpu_time);
	
	free(a); free(b); free(c);
	hipFree(da); hipFree(db), hipFree(dc);

	return 0;
}


