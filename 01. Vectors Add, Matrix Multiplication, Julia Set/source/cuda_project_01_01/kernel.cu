#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "opencv\cv.h"
#include "opencv\highgui.h"

//#define NUM_DATA 123456
//using namespace cv;

__global__ void vec_add(float *a, float *b, float *c, int n){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid >= n) return;
	c[tid] = a[tid] + b[tid];
}

int main(){
	// �ǽ� 1-1
	int data, thread; // data:�� �������� ��, thread:�������� ����
	hipEvent_t start,end;
	float gpu_time;
	printf("�� data�� ������ ���� thread ������ �Է��Ͻÿ�(ex. 123456, 1024) : ");
	scanf("%d, %d", &data, &thread);

	hipEventCreate(&start);
	hipEventCreate(&end);

	float *a,*b,*c;
	a = (float *)malloc(sizeof(float) * data);
	b = (float *)malloc(sizeof(float) * data);
	c = (float *)malloc(sizeof(float) * data);

	for(int i = 0 ; i < data ; i++){
		a[i] = i*2; // ������ ��
		b[i] = i+3; // ������ ��
	}

	int block_num = data / thread;

	float *da, *db, *dc;
	hipMalloc((void**)&da, sizeof(float)*data);
	hipMalloc((void**)&db, sizeof(float)*data);
	hipMalloc((void**)&dc, sizeof(float)*data);

	hipMemcpy(da,a,sizeof(float)*data,hipMemcpyHostToDevice);
	hipMemcpy(db,b,sizeof(float)*data,hipMemcpyHostToDevice);
	
	hipEventRecord(start, 0);
	vec_add<<<block_num,thread>>>(da,db,dc,data);
	// block_num : �� ����
	// thread : ���� ������ ����
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&gpu_time, start, end); // �� ���� �ð��� �󸶳� �귶����
	
	hipMemcpy(c, dc, sizeof(float)*data, hipMemcpyDeviceToHost);

	printf("\n");
	printf("gpu time = %f\n", gpu_time);

	free(a);free(b);free(c);
	hipFree(da);hipFree(db);hipFree(dc);
	
	return 0;
}

