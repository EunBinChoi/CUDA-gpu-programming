#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <windows.h>

#include "opencv\cv.h"
#include "opencv\highgui.h"

__global__ void mat_mul(float *a, float *b, float *c, int M, int N, int K){
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	float sum = 0;

	for(int k = 0; k < K; k++){
		sum += a[row*K+k] * b[k*N+col];
	}

	c[row*N + col] = sum;
}

void nmat_mul(float *a, float *b, float *c, int M, int N, int K){

	float sum = 0;

	for(int i = 0 ; i < M ; i++) {
		for(int j = 0 ; j < N ; j++) {
			for(int k = 0; k < K; k++) {
				sum += a[i*K+k] * b[k*N+j];
			}
			c[i*N+j] = sum;
			sum = 0;
		}
	}
}

int main(){

	// �ǽ� 2-3(�������� ������ 2,4�� �� �۵����� ����)
	hipEvent_t start, end;
	float gpu_time;
	
	hipEventCreate(&start);
	hipEventCreate(&end);

	const int M = 2048;
	const int N = 1536;
	const int K = 1024;

	float *a, *b, *c;
	a = (float *)malloc(sizeof(float) * M*K);
	b = (float *)malloc(sizeof(float) * K*N);
	c = (float *)malloc(sizeof(float) * M*N);

	for(int i = 0; i < M*K ; i++)
		a[i] = (float)(rand() - RAND_MAX/2)/(RAND_MAX/2); 
	for(int i = 0; i < K*N ; i++)
		b[i] = (float)(rand() - RAND_MAX/2)/(RAND_MAX/2);

	float *da, *db, *dc;
	hipMalloc((void**)&da, sizeof(float) * M*K);
	hipMalloc((void**)&db, sizeof(float) * K*N);
	hipMalloc((void**)&dc, sizeof(float) * M*N);

	hipMemcpy(da, a, sizeof(float)*M*K,hipMemcpyHostToDevice);
	hipMemcpy(da, b, sizeof(float)*K*N,hipMemcpyHostToDevice);

	int thread;
	//2,4,6,8,12,16,24,32

	//i�� 2,4�϶� �۵����� ����
	for(int i = 4 ; i <= 32 ; ){
		Sleep(1000);
		gpu_time = 0;
		thread = i;
		
		hipEventRecord(start, 0);
		// 2���� �迭 => dim3
		dim3 dim_blocks(thread, thread, 1);
		dim3 dim_grid(N/thread, M/thread, 1);
		mat_mul<<<dim_grid, dim_blocks>>>(da,db,dc,M,N,K);
		
		hipEventRecord(end, 0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&gpu_time, start, end);
		//printf("%f\n", gpu_time);
		hipMemcpy(c, dc, sizeof(float)*M*N,hipMemcpyDeviceToHost);

		printf("thread = %d, gpu time = %.10f\n",thread, gpu_time);

		if(i < 8) i = i + 2;
		else if(i < 16) i = i + 4;
		else if(i < 32) i = i + 8;
		else break;
	}

	free(a); free(b); free(c);
	hipFree(da); hipFree(db); hipFree(dc);

	return 0;

}

