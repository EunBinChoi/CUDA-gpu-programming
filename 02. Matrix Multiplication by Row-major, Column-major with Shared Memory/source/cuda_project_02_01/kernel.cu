#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <windows.h>

#include "opencv\cv.h"
#include "opencv\highgui.h"

__global__ void mat_mul(float *a, float *b, float *c, int M, int N, int K){
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	float sum = 0;

	for(int k = 0; k < K; k++){
		sum += a[row*K+k] * b[k*N+col];
	}

	c[row*N + col] = sum;
}

void nmat_mul(float *a, float *b, float *c, int M, int N, int K){

	float sum = 0;

	for(int i = 0 ; i < M ; i++) {
		for(int j = 0 ; j < N ; j++) {
			for(int k = 0; k < K; k++) {
				sum += a[i*K+k] * b[k*N+j];
			}

			c[i*N+j] = sum;
			sum = 0;
		}
	}
}


int main(){
	hipEvent_t start, end;
	float gpu_time;
	
	hipEventCreate(&start);
	hipEventCreate(&end);

	const int M = 2048;
	const int N = 1536;
	const int K = 1024;

	float *a, *b, *c;
	a = (float *)malloc(sizeof(float) * M*K);
	b = (float *)malloc(sizeof(float) * K*N);
	c = (float *)malloc(sizeof(float) * M*N);

	for(int i = 0; i < M*K ; i++)
		a[i] = (float)(rand() - RAND_MAX/2)/(RAND_MAX/2); 
	for(int i = 0; i < K*N ; i++)
		b[i] = (float)(rand() - RAND_MAX/2)/(RAND_MAX/2);

	float *da, *db, *dc;
	hipMalloc((void**)&da, sizeof(float) * M*K);
	hipMalloc((void**)&db, sizeof(float) * K*N);
	hipMalloc((void**)&dc, sizeof(float) * M*N);

	hipMemcpy(da, a, sizeof(float)*M*K,hipMemcpyHostToDevice);
	hipMemcpy(da, b, sizeof(float)*K*N,hipMemcpyHostToDevice);

	int thread = 16;

	for(int i = 1 ; i <= 2 ; i++){
		Sleep(1000);
		gpu_time = 0;
		thread = 16*i;
		
		hipEventRecord(start, 0);
		dim3 dim_blocks(thread, thread, 1);
		dim3 dim_grid(N/thread, M/thread, 1);
		
		mat_mul<<<dim_grid, dim_blocks>>>(da,db,dc,M,N,K);
	
		hipEventRecord(end, 0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&gpu_time, start, end);

		hipMemcpy(c, dc, sizeof(float)*M*N,hipMemcpyDeviceToHost);

		printf("thread = %d, gpu time = %.10f\n",thread, gpu_time);
	}

	free(a); free(b); free(c);
	hipFree(da); hipFree(db); hipFree(dc);

	return 0;

}

