#include "hip/hip_runtime.h"
// ���� �޸𸮸� �̿��� ��İ��� Ŀ���� �ۼ��Ͻÿ�
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "opencv\cv.h"
#include "opencv\highgui.h"

#define TILE_DIM 32
//#define BLOCK_ROWS 16
#define BLOCK_COLS 32 // ���⼭ block_cols�� tile_dim �� ���ٰ� ����

__global__ void gpu_transpose(int*a, int*b, int M, int N){
   __shared__ int tile[TILE_DIM][TILE_DIM];
   __shared__ int tile_result[TILE_DIM][TILE_DIM];

   int x = blockIdx.x * TILE_DIM + threadIdx.x;
   int y = blockIdx.y * TILE_DIM + threadIdx.y;
   
   int width = gridDim.x * TILE_DIM;
   int height = gridDim.y * TILE_DIM;

   //// ���� § �ҽ� (����1) // �� ������ block_cols�� tile_dim���� �۾Ƶ� ������(���ÿ� ����Ǵ� ������ cols�� ���� < Ÿ�� ����)
   //for (int j = 0; j < TILE_DIM ; j += BLOCK_COLS) // ���� ������ �Ѿ�� ����
   //   tile[threadIdx.y][threadIdx.x + j] = a[(y * width) + x + j];

   //__syncthreads();
   //      
   //x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
   //y = blockIdx.x * TILE_DIM + threadIdx.y;

   //for (int j = 0; j < TILE_DIM; j += BLOCK_COLS){
   //   b[(y * height) + x + j] = tile[threadIdx.x + j][threadIdx.y];
   //}
   //__syncthreads();

   // �����Բ��� § �ҽ� (����2)
   //for (int j = 0; j < TILE_DIM ; j += BLOCK_COLS)
   
   tile[threadIdx.y][threadIdx.x] = a[(y*width) + x];

   __syncthreads();
         
   x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
   y = blockIdx.x * TILE_DIM + threadIdx.y;

   //for (int j = 0; j < TILE_DIM; j += BLOCK_COLS){
      b[(y*height) + x] = tile[threadIdx.x][threadIdx.y];
  // }
   __syncthreads();
}

int main(){
   hipEvent_t start, end;
   float gpu_time; 

   hipEventCreate(&start);
   hipEventCreate(&end);

	const int M = 2048;
	const int N = 1024;
   //const int K = 1024;

   int *shared_a;
   int *shared_b;
   shared_a = (int *)malloc(sizeof(int) * M*N);
   shared_b = (int *)malloc(sizeof(int) * M*N);

   for(int i =0 ; i < M*N; i++) shared_a[i] = (int)(rand()%100);
   for(int i =0 ; i < M*N; i++) shared_b[i] = 0;

   int *shared_da;
   int *shared_db;

   hipMalloc((void**)&shared_da, sizeof(int) * M*N);
   hipMalloc((void**)&shared_db, sizeof(int) * M*N);
   hipMemcpy(shared_da, shared_a, sizeof(int)*M*N,hipMemcpyHostToDevice);

   int thread = 32;

   hipEventRecord(start, 0);
   // ���� 1
   //dim3 dim_block(BLOCK_COLS, thread , 1);
   // ���� 2
   dim3 dim_block(thread,thread , 1);
   dim3 dim_grid(N/thread, M/thread, 1);
   //mat_mul<<<dim_grid, dim_block>>>(da,db,dc,M,N,K);
   gpu_transpose<<<dim_grid, dim_block>>>(shared_da,shared_db,M,N);

   hipEventRecord(end, 0);
   hipEventSynchronize(end);
   hipEventElapsedTime(&gpu_time, start, end); // �ð� ������ ��

   //hipMemcpy(c, dc, sizeof(float)*M*N,hipMemcpyDeviceToHost);
   hipMemcpy(shared_b, shared_db, sizeof(int)*M*N,hipMemcpyDeviceToHost);

   printf("���� thread ���� = %d, shared gpu time = %f\n", thread, gpu_time);

   //for(int i = 0 ; i < M ; i++){
   //   for(int j = 0 ; j < N ; j++)
   //      printf("%10f ", shared_c[i*N+j]);
   //   printf("\n");
   //}
   int i = 0;
   int j = 0;

   //for(i = 0 ; i < M ; i++){
   //   for(j = 0; j < N ; j++){
   //      printf("%4d ",shared_a[i*N+j]);        
   //   }
   //   printf("\n");
   //}
   //printf("\n");printf("\n");
   //
   //for(i = 0 ; i < N ; i++){
   //   for(j = 0; j < M ; j++){
   //      printf("%4d ",shared_b[i*M+j]);        
   //   }
   //   printf("\n");
   //}
   // 
   printf("\n");
   for(i = 0 ; i < M ; i++){
      for(j = 0; j < N ; j++){
         // printf("%4d ",shared_a[i*N+j]);
         if(abs(shared_a[i*N+j]-shared_b[j*M+i]) > 0.0001){
            printf("=>[ false ]\n");
            //  printf("%d %d %d %d\n",i,j ,shared_a[i*N+j],shared_b[j*M+i]);
            return 0;
         }
      }
      //printf("\n");
   }
   if(i*j == M*N) printf("=>[ ��� ��ġ�մϴ� ]\n");
  
   
   free(shared_a); free(shared_b);
   hipFree(shared_da); hipFree(shared_db);
   
   return 0;
}