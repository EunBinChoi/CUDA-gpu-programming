#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "opencv\cv.h"
#include "opencv\highgui.h"

__global__ void gpu_transpose(int *a, int* b, int M, int N){	

	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	b[col*M + row] = a[row*N + col];
}

int main(){
	hipEvent_t start, end;
	float gpu_time; 
	
	hipEventCreate(&start);
	hipEventCreate(&end);

	const int M = 2048;
	const int N = 1024;
	
	int *a, int* b;
	a = (int *)malloc(sizeof(int) * M*N);
	b = (int *)malloc(sizeof(int) * M*N);

	for(int i =0 ; i < M*N; i++) 
		a[i] = (int)(rand()%100);

	int *da,*db;
	hipMalloc((void**)&da, sizeof(int) * M*N);
	hipMalloc((void**)&db, sizeof(int) * M*N);
	
	hipMemcpy(da, a, sizeof(int)*M*N,hipMemcpyHostToDevice);
	//hipMemcpy(db, b, sizeof(float)*M*N,hipMemcpyHostToDevice);
	
	int thread = 32;
	hipEventRecord(start, 0);
	dim3 dim_block(thread, thread, 1);
	dim3 dim_grid(N/thread, M/thread, 1);
	gpu_transpose<<<dim_grid, dim_block>>>(da,db,M,N);

	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&gpu_time, start, end); // �ð� ������ ��

	
	hipMemcpy(b, db, sizeof(int)*M*N,hipMemcpyDeviceToHost);
	
	printf("���� thread ���� = %d, gpu time = %f\n", thread, gpu_time);
	
	int i = 0, j = 0;

	printf("\n");
	for(i = 0 ; i < M ; i++){
		for(j = 0; j < N ; j++){
			if(abs(a[i*N+j]-b[j*M+i]) > 0.0001){
				printf("=>[ false ]\n");
				return 0;
			}
		}
	}
	if(i*j == M*N) printf("=>[ ��� ��ġ�մϴ� ]\n");
	return 0;
}


