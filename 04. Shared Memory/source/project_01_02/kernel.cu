#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#define SHARED 1024

__global__ void reduce(int*a ,int *o, int n)
{
	__shared__ int sa[SHARED]; // �����޸�
	
	int tid = threadIdx.x;
	int idx = blockIdx.x * blockDim.x+ threadIdx.x;
	/*
	if(idx > n){
		if(threadIdx.x < SHARED)
		{
			sa[tid] = 0;
			return;
		}
	}*/
	if(idx + (int)n/2 >= n) sa[tid] = a[idx];
	if((n/2) % blockDim.x  == 0) sa[tid] = a[idx] + a[idx + (int)n/2];
	// ������������� ����
	
	__syncthreads();
	
	// ok
	for(int s = blockDim.x/2; s >= 1; s >>= 1){
		if(tid < s)
			sa[tid] = sa[tid] + sa[tid + s];
		__syncthreads();
	}
	
	if(tid == 0) o[blockIdx.x] = sa[tid];
}

int main(){
	const int N = 1000*1024;
	int block_size = 1024;
	int block_num = (N + block_size -1)/block_size;
	
	int *a, *o;
	a = (int *)malloc(sizeof(int) * N);
	o = (int *)malloc(sizeof(int) * block_num);
	for(int i = 0 ; i < N; i++) {
		a[i] = (i % 2 ==0) ? i : -i;
		//printf("%d ", a[i]);
	}

	int *id, *od;
	hipMalloc((void **)&id, sizeof(int)*N);
	hipMalloc((void **)&od, sizeof(int)*block_num);
	
	hipEvent_t start, end;
	float gpu_time;
	hipEventCreate(&start);
	hipEventCreate(&end);

	hipMemcpy(id,a,sizeof(int)*N, hipMemcpyHostToDevice);

	hipEventRecord(start, 0);
	reduce<<<block_num/2, block_size>>>(id,od,N);
	
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&gpu_time, start, end);
	
	printf("\n>>gpu_time = %f\n", gpu_time);
	//printf("%d\n\n", N/2);
	hipMemcpy(o,od,sizeof(int)*block_num, hipMemcpyDeviceToHost);
	int sum = 0;
	for(int i = 0 ; i < block_num; i++){
		//printf("%d ", o[i]);
		sum += o[i];
	}
	printf("\n");
	printf("GPU : %d", sum);
	printf("\n");

	//CPU
	int sum2 = 0;
	for(int i = 0 ; i < N ; i ++)
	{
		sum2 += a[i];
	}
	
	printf("CPU : %d\n",sum2);

	hipError_t err;
	err = hipGetLastError();
	printf("%s\n", hipGetErrorString(err));

	free(a); free(o);
	hipFree(id); hipFree(od);

	return 0;
}