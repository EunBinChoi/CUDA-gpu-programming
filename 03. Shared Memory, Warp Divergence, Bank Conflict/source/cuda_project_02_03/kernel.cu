#include "hip/hip_runtime.h"
// reduce 2
// ���� �б� �ذ�
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//// ���� 1(�߸��� �˰���)
//__global__ void reduce(int*a, int*b, int*c, int*o, int n){
//	__shared__ int sa[1024];
//	__shared__ int sb[1024];
//	__shared__ int sc[1024];
//
//	int tid = threadIdx.x;
//	int idx = blockIdx.x * blockDim.x + threadIdx.x;
//	int tmp = tid * 2;
//	
//	sa[tid] = a[idx];
//	sb[tid] = b[idx];
//	sc[tid] = c[idx];	
//	__syncthreads();
//
//	
//	sc[tmp] = (sa[tmp] * sb[tmp]) + (sa[tmp + 1] * sb[tmp + 1]);
//	__syncthreads();
//		
//
//	for(int s = 1; s < blockDim.x ; s *= 2){
//		int index = tid * s * 2;
//		if(index < blockDim.x)
//			 sc[index] = sc[index] + sc[index + s];
//		__syncthreads();
//	}
//	
//
//	if(tid == 0) o[blockIdx.x] = sc[tid];
//
//}


// ���� 2
__global__ void reduce(int*a, int*b, int*c, int*o, int n){
	__shared__ int sa[1024];
	__shared__ int sb[1024];
	__shared__ int sc[1024];

	int tid = threadIdx.x;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//if(tid >= n) return;
	// �����޸𸮰� ũ�� ������ �����Ƿ� �ʿ� ����

	sa[tid] = a[idx];
	sb[tid] = b[idx];
	sc[tid] = c[idx];	
	__syncthreads();

	sc[tid] = (sa[tid] * sb[tid]);
	__syncthreads();
		
	for(int s = 1; s < blockDim.x ; s *= 2){
		int index = tid * s * 2;
		if((index < blockDim.x))
			 sc[index] = sc[index] + sc[index + s];
		__syncthreads();
	}
	if(tid == 0) o[blockIdx.x] = sc[tid];
}


int main(){
	
	// ���� ����
	FILE *fp1, *fp2;
	char file_name1[50] = "seta.dat";
	char file_name2[50] = "setb.dat";

	int *matrix_a ,int *matrix_b;
	
	fp1 = fopen(file_name1, "r");
	if(fp1 == NULL){
		printf("File Open Error");
		return;
	}

	int num1;
	fscanf(fp1, "%d", &num1);
	
	fp2 = fopen(file_name2, "r");
	if(fp2 == NULL){
		printf("File Open Error");
		return;
	}
	
	int num2;
	fscanf(fp2, "%d", &num2);
	
	matrix_a = (int *)malloc(sizeof(int) * num1);
	matrix_b = (int *)malloc(sizeof(int) * num2);
	
	for(int i = 0 ; i < num1 ; i++){
		fscanf(fp1, "%d", &(matrix_a[i]));
	}
	printf("\n");

	for(int i = 0 ; i < num2 ; i++){
		fscanf(fp2, "%d",&(matrix_b[i]));
	}
	printf("\n");
	
	fclose(fp1);
	fclose(fp2);
	
		
	// ���� ����
	//const int N = 1000 * 1024;
	int block_size = 1024;
	// �� ������� 2 �̻��̾���Ѵ�
	int block_num = (num1 + block_size -1) / block_size;
	//printf(">>%d",block_num);
	int *o = (int *)malloc(sizeof(int) * block_num);

	int *da, *db, *dc, *dd;
	hipMalloc((void **)&da, sizeof(int)*num1);
	hipMalloc((void **)&db, sizeof(int)*num1);
	hipMalloc((void **)&dc, sizeof(int)*num1);
	hipMalloc((void **)&dd, sizeof(int)*block_num);
	
	hipEvent_t start, end;
	float gpu_time;
	hipEventCreate(&start);
	hipEventCreate(&end);

	hipMemcpy(da,matrix_a,sizeof(int)*num1, hipMemcpyHostToDevice);
	hipMemcpy(db,matrix_b,sizeof(int)*num1, hipMemcpyHostToDevice);
	
	hipEventRecord(start, 0);
	reduce<<<block_num, block_size>>>(da,db,dc,dd,num1);
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&gpu_time, start, end);
	
	hipMemcpy(o,dd,sizeof(int)*block_num, hipMemcpyDeviceToHost);
	//hipMemcpy(o,dd,sizeof(int)*block_num, hipMemcpyDeviceToHost);

	printf("\n\n");
	printf("<< ����� >> \n");
	int sum = 0;
	for(int i = 0 ; i < block_num ; i ++){
		//printf("[index = %d] : %7d\n",i,o[i]);
		sum += o[i];
	}
	printf("\n\n");
	printf(">> sum : %d", sum);
	printf("\n\n");
	printf("<< ����ð� >>\ngpu_time = %f\n", gpu_time);

	int result = 0;
	for(int i = 0; i < num1; i++){
		result += matrix_a[i] * matrix_b[i];
		//printf("%d\n",result);
	}
	printf("%d\n", result);
	return 0;
}