// reduce 2 ���α׷� ����
// reduce2 => ���� �б� �ذ�
// 0 2 4 ... => ���ϴ� thread, 1 3 5 ... => ��� thread
// ���� �бⰡ �Ͼ�� �ʵ��� �Ѵ�

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void reduce(int*a, int*o, int n){
	__shared__ int sa[1024]; // �����޸�
	int tid = threadIdx.x;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	sa[tid] = a[idx];
	__syncthreads();

	for(int s = 1; s < blockDim.x; s *= 2){
		int index = tid*s*2;
		
		if(index < blockDim.x)
		{
			if(sa[index] < sa[index + s]) sa[index] = sa[index];
			else						  sa[index] = sa[index + s];
		}
		__syncthreads();
	}

	if(tid == 0) o[blockIdx.x] = sa[tid]; 
}
int main(){
	
	const int N			= 1000*1024;
	int block_size		= 1024;
	int block_num		= (N + block_size -1)/block_size;

	int *a, *o;
	a = (int *)malloc(sizeof(int) * N);
	o = (int *)malloc(sizeof(int) * block_num);
	
	for(int i = 0 ; i < N ; i++){
		a[i] = (i%2 == 0)? i : -i;
	}
	int *id, *od;
	hipMalloc((void **)&id, sizeof(int)*N);
	hipMalloc((void **)&od, sizeof(int)*block_num);
	
	hipEvent_t start, end;
	float gpu_time;
	hipEventCreate(&start);
	hipEventCreate(&end);

	hipMemcpy(id,a,sizeof(int)*N, hipMemcpyHostToDevice);

	hipEventRecord(start, 0);
	reduce<<<block_num, block_size>>>(id,od,N);
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&gpu_time, start, end);
	
	printf("\n>>gpu_time = %f\n", gpu_time);
	
	hipMemcpy(o,od,sizeof(int)*block_num, hipMemcpyDeviceToHost);
	
	int min = INT_MAX;
	for(int i = 0 ; i < block_num ; i++){
		if(min > o[i]) min = o[i];
	}
	printf("gpu �ּҰ� = %d\n", min);
	
	/*for(int i = 0 ; i < block_num ; i ++){
		printf("%d ", o[i]);
	}*/
	printf("\n");

	//----------------------------------------------
	// cpu���� �ּҰ� ���ϱ�
	int min_cpu = INT_MAX;
	for(int i = 0 ; i < N ; i ++)
	{
		if(a[i] < min_cpu) 
			min_cpu = a[i];
	}
	
	printf("cpu �ּҰ� = %d", min_cpu);

	free(a); free(o);
	hipFree(id); hipFree(od);

	return 0;
}